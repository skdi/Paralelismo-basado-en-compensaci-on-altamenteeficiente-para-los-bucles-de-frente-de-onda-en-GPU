//A CUDA based implementation of the Smith Waterman Algorithm
//Author: Romil Bhardwaj

#include "hip/hip_runtime.h"

#include<time.h>

#include <stdio.h>
#include <stdlib.h>
#define max(a,b) (((a)>(b))?(a):(b))

//Define the costs here
#define indel -1
#define match 2
#define mismatch -1
#define TILE_WIDTH 8
#define NUM_THREADS 32

//CHANGE THIS VALUE TO CHANGE THE NUMBER OF ELEMENTS
const int arraySize = 65536;
#define Width arraySize+1
int SIZE = arraySize+1 * arraySize+1;
int SIZED = SIZE * sizeof(int);
//CHANGE THIS VALUE TO CHANGE THE NUMBER OF ELEMENTS

hipError_t SWHelper(int (*c)[arraySize+1], const char *a, const char *b, size_t size);
hipError_t SWHelperL(int* c, const char *a, const char *b, size_t size);


__global__ void SmithWKernelExpand(int (*c)[arraySize+1], const char *a, const char *b, const int *k)		//Declared consts to increase access speed
{
    int i = threadIdx.x+1;
	int j = ((*k)-i)+1;
	int north=c[i][(j)-1]+indel;			//Indel
	int west=c[i-1][j]+indel;
	int northwest;
	if (((int) a[i-1])==((int)b[(j)-1]))
		northwest=c[i-1][(j)-1]+match;		//Match
	else
		northwest=c[i-1][(j)-1]+mismatch;		//Mismatch
    c[i][j] = max(max(north, west),max(northwest,0));
	//c[i][j]=(*k);						//Debugging - Print the antidiag num
}

__global__ void SmithWKernelShrink(int (*c)[arraySize+1], const char *a, const char *b, const int *k)
{
    int i = threadIdx.x+((*k)-arraySize)+1;
	int j = ((*k)-i)+1;
	int north=c[i][(j)-1]+indel;			//Indel
	int west=c[i-1][j]+indel;
	int northwest;
	if (((int) a[i-1])==((int)b[(j)-1]))
		northwest=c[i-1][(j)-1]+match;		//Match
	else
		northwest=c[i-1][(j)-1]+mismatch;		//Mismatch
    c[i][j] = max(max(north, west),max(northwest,0));
	//c[i][j]=(*k);						//Debugging - Print the antidiag num
}


__global__ void SmithWKernelExpandL(int *c, const char *a, const char *b, const int *k)		//Declared consts to increase access speed
{
  int i = threadIdx.x+1;
	int j = ((*k)-i)+1;
	int north=c[i*(arraySize+1)+(j)-1]+indel;			//Indel
	int west=c[i*(arraySize+1)-1+j]+indel;
	int northwest;
	if (((int) a[i-1])==((int)b[(j)-1]))
		northwest=c[i*(arraySize+1)-1+(j)-1]+match;		//Match
	else
		northwest=c[i-1+(j)-1]+mismatch;		//Mismatch
    c[i*(arraySize+1)+j] = max(max(north, west),max(northwest,0));	
	//c[i][j]=(*k);						//Debugging - Print the antidiag num
}

__global__ void SmithWKernelShrinkL(int *c, const char *a, const char *b, const int *k)
{
  int i = threadIdx.x+((*k)-arraySize)+1;
	int j = ((*k)-i)+1;
	int north=c[i*(arraySize+1)+(j)-1]+indel;			//Indel
	int west=c[i*(arraySize+1)-1+j]+indel;
	int northwest;
	if (((int) a[i-1])==((int)b[(j)-1]))
		northwest=c[i*(arraySize+1)-1+(j)-1]+match;		//Match
	else
		northwest=c[i*(arraySize+1)-1+(j)-1]+mismatch;		//Mismatch
    c[i*(arraySize+1)+j] = max(max(north, west),max(northwest,0));
	//c[i][j]=(*k);						//Debugging - Print the antidiag num
}

void print(int c[arraySize+1][arraySize+1]){
	int j=0,i=0;
	for (i = 0; i < arraySize+1; i++) {
        for (j = 0; j < arraySize+1; j++) {
            printf("%d \t", c[i][j]);
        }
        printf("\n");
	}
}
void printL(int *c){
	int j=0,i=0;
	for (i = 0; i < arraySize+1; i++) {
        for (j = 0; j < arraySize+1; j++) {
            printf("%d \t", c[i*arraySize+1 + j]);
        }
        printf("\n");
	}
}

//matriz de entrada, i y j (salida) posicion de mayor valor
__global__ void MaximosTiled(int *c,int &i,int &j)
{
	__shared__ int sub_matriz[TILE_WIDTH][TILE_WIDTH];
	//__shared__ int Nds[TILE_WIDTH][TILE_WIDTH];

	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;
	int max_local = 0;

	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;

	int Pvalue = 0;
	//Row*Width + ph*TILE_WIDTH + tx
	for (int ph = 0; ph < Width/TILE_WIDTH; ++ph) {
	//Cargando los datos en las submatrices puestas en memoria compartida
		if ((Row< Width) && (ph*TILE_WIDTH+tx)< Width && ((ph*TILE_WIDTH+ty)<Width && Col<Width))
			sub_matriz[ty][tx] = c[Row*Width + ph*TILE_WIDTH + tx];
		__syncthreads();
	
		//Multiplicando las submatrices 
		if(max_local < sub_matriz[tx][ty]){
			max_local = sub_matriz[tx][ty];
			i=Row*Width;
			j=ph*TILE_WIDTH + tx;
		}

		}

	
}

void traceback_tiled(int *c, char a[], char b[]){
	int j=0,i=0;
	int maxi=0,maxj=0,max=0;
	
	int *c_d;
	//Separando memoria para la matriz en el device
	hipMalloc((void **)&c_d, SIZED);
	//Cargando la data c en el device
	hipMemcpy(c_d, c, SIZED, hipMemcpyHostToDevice);
	//Definiendo 
	//Numero de threads por bloque
	dim3 threadsPerBlock(32,32);
	//Numero de bloques por SM
	dim3 blocksPerGrid(800,800);
	threadsPerBlock.x = NUM_THREADS;
	threadsPerBlock.y = NUM_THREADS;
	blocksPerGrid.x = ceil(double(Width) / double(threadsPerBlock.x));
	blocksPerGrid.y = ceil(double(Width) / double(threadsPerBlock.y));

	MaximosTiled <<<blocksPerGrid, threadsPerBlock>>>(c_d,maxi,maxj);
	hipFree(c_d);
	i=maxi;
	j=maxj;
	printf("The optimal local alignment starts at index %d for a, and index %d for b.\n", i,j);
	while (c[i*Width+j]!=0 && i>=0 && j>=0 ){
		printf("\n");
		if (c[i*Width+j]==c[(i-1)*Width+(j)-1]+match){		//From match
			i--;
			j--;
			printf("%c -- %c", a[i], b[j]);
		}
		else if (c[i*Width+j]==c[i-1*Width+(j)-1]+mismatch){ //From mismatch
			i--;
			j--;
			printf("%c -- %c", a[i], b[j]);
		}
		else if (c[i*Width+j]==c[i*Width+(j)-1]+indel){	//North
			j--;
			printf("- -- %c", b[j]);
		}
		else{									//Else has to be from West
			i--;
			printf("%c -- -", a[i]);
		}
	}
	
	printf("\n\nThe optimal local alignment ends at index %d for a, and index %d for b.\n", i,j);
}


void traceback(int c[arraySize+1][arraySize+1], char a[], char b[]){
	int j=0,i=0;
	int maxi=0,maxj=0,max=0;
	for (i = 0; i < arraySize+1; i++) {
        for (j = 0; j < arraySize+1; j++) {
           if(c[i][j]>max){
			   maxi=i;
			   maxj=j;
				max=c[i][j];
		   }
        }
	}
	i=maxi;
	j=maxj;
	printf("The optimal local alignment starts at index %d for a, and index %d for b.\n", i,j);
	while (c[i][j]!=0 && i>=0 && j>=0 ){
		printf("\n");
		if (c[i][j]==c[i-1][(j)-1]+match){		//From match
			i--;
			j--;
			printf("%c -- %c", a[i], b[j]);
		}
		else if (c[i][j]==c[i-1][(j)-1]+mismatch){ //From mismatch
			i--;
			j--;
			printf("%c -- %c", a[i], b[j]);
		}
		else if (c[i][j]==c[i][(j)-1]+indel){	//North
			j--;
			printf("- -- %c", b[j]);
		}
		else{									//Else has to be from West
			i--;
			printf("%c -- -", a[i]);
		}
	}
	
	printf("\n\nThe optimal local alignment ends at index %d for a, and index %d for b.\n", i,j);
}


int main()
{
	char b[arraySize];//{'a','c','a','c','a','c','t','a'};
	char a[arraySize];//{'a','g','c','a','c','a','c','a'};
	
	int i=0;
	
	//Generating the sequences:
	
	srand (time(NULL));
	printf("\nString a is: ");
    for(i=0;i<arraySize;i++)
    {
        int gen1=rand()%4;
        switch(gen1)
        {
            case 0:a[i]='a';
            break;
            case 1: a[i]='c';
            break;
            case 2: a[i]='g';
            break;
            case 3: a[i]='t';
        }
		//a[i]='a';
		printf("%c ", a[i]);
    }

	printf("\nString b is: ");
	for(i=0;i<arraySize;i++)
    {
        int gen1=rand()%4;
        switch(gen1)
        {
            case 0:b[i]='a';
            break;
            case 1: b[i]='c';
            break;
            case 2: b[i]='g';
            break;
            case 3: b[i]='t';
        }
		//b[i]='a';
		printf("%c ", b[i]);
    }
	
	
	printf("\nOkay, generated the string \n");
	int c[arraySize+1][arraySize+1] = { {0} };
	int *h_c = (int *)malloc(SIZED);

	clock_t start=clock();

    // Run the SW Helper function
    hipError_t cudaStatus = SWHelper(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "SWHelper failed!");
        return 1;
    }
	
	clock_t end=clock();
	print(c);


	//cudaError_t cudaStado= SWHelperL(h_c,a,b,arraySize);
	//print(c);


	//Printing the final score matrix. Uncomment this to see the matrix.
	//print(c);

	
    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	traceback_tiled(h_c,a,b);
	printf("\n\nEnter any number to exit.");
	printf("\n\nTotal time taken is %f seconds\n",(double)(end-start)/CLOCKS_PER_SEC);
	int x;
	scanf("%d", &x);
    return 0;
}

// Helper function for SmithWaterman
hipError_t SWHelper(int (*c)[arraySize+1], const char *a, const char *b, size_t size)
{
    char *dev_a;
    char *dev_b;
	int (*dev_c)[arraySize+1] = {0};
	int (*j)=0;
	int *dev_j;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
       // goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, (size+1) * (size+1) * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
       // goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        //goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        //goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_j, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        //goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        //goto Error;
    }

	cudaStatus = hipMemcpy(dev_j, &j, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        //goto Error;
    }


    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        //goto Error;
    }

	cudaStatus = hipMemcpy(dev_c, c, (size+1) * (size+1) * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        //goto Error;
    }

	int i=0;
	clock_t start1=clock();

    // Launch a kernel on the GPU with one thread for each element.

	//Expanding Phase
	for (i=1; i<size+1; i++){
		cudaStatus = hipMemcpy(dev_j, &i, sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!", cudaStatus);
			//goto Error;
		}
		SmithWKernelExpand<<<1, i>>>(dev_c, dev_a, dev_b, dev_j);
	}

	//Shrink Phase
	for (int k=size-1; k>0; k--, i++){
		cudaStatus = hipMemcpy(dev_j, &i, sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			//goto Error;
		}

		SmithWKernelShrink<<<1, k>>>(dev_c, dev_a, dev_b, dev_j);
	}
	clock_t end1=clock();
    printf("\n\nKernel Time taken is %f seconds\n",(double)(end1-start1)/CLOCKS_PER_SEC);


    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching SmithWKernel!\n", cudaStatus);
  //      goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
	//cudaStatus = cudaMemcpy2D(c,size * size * sizeof(int),dev_c,size * size * sizeof(int),size * size * sizeof(int),size * size * sizeof(int),cudaMemcpyDeviceToHost);
    cudaStatus = hipMemcpy(c, dev_c, (size+1) * (size+1) * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
  //      goto Error;
    }

//Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
} 

hipError_t SWHelperL(int* c, const char *a, const char *b, size_t size)
{
    char *dev_a;
    char *dev_b;
	int (*dev_c);
	int (*j)=0;
	int *dev_j;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
       // goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, (size+1) * (size+1) * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
       // goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        //goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        //goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_j, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        //goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        //goto Error;
    }

	cudaStatus = hipMemcpy(dev_j, &j, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        //goto Error;
    }


    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        //goto Error;
    }

	cudaStatus = hipMemcpy(dev_c, c, (size+1) * (size+1) * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        //goto Error;
    }

	int i=0;
	clock_t start1=clock();

    // Launch a kernel on the GPU with one thread for each element.

	//Expanding Phase
	for (i=1; i<size+1; i++){
		cudaStatus = hipMemcpy(dev_j, &i, sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!", cudaStatus);
			//goto Error;
		}
		SmithWKernelExpandL<<<1, i>>>(dev_c, dev_a, dev_b, dev_j);
	}

	//Shrink Phase
	for (int k=size-1; k>0; k--, i++){
		cudaStatus = hipMemcpy(dev_j, &i, sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			//goto Error;
		}

		SmithWKernelShrinkL<<<1, k>>>(dev_c, dev_a, dev_b, dev_j);
	}
	clock_t end1=clock();
    printf("\n\nKernel Time taken is %f seconds\n",(double)(end1-start1)/CLOCKS_PER_SEC);


    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching SmithWKernel!\n", cudaStatus);
  //      goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
	//cudaStatus = cudaMemcpy2D(c,size * size * sizeof(int),dev_c,size * size * sizeof(int),size * size * sizeof(int),size * size * sizeof(int),cudaMemcpyDeviceToHost);
    cudaStatus = hipMemcpy(c, dev_c, (size+1) * (size+1) * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
  //      goto Error;
    }

//Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
} 
